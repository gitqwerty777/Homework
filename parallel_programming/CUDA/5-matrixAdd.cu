#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MAXN 1024
int arr[MAXN];

__global__ void add(int A[MAXN][MAXN], int B[MAXN][MAXN]){
  int i = blockDim.x*blockIdx.x+threadIdx.x+1;
  int j = blockDim.y*blockIdx.y+threadIdx.y+1;
  int li = threadIdx.x+1;
  int lj = threadIdx.y+1;
  __shared__ unsigned int sur[BSIDE+2][BSIDE+2];//surroundings

  sur[li][lj] = arr[i][j];
  
  
}

int main() {
  int n;
  uint32_t key;

  dim3 grid=dim3(NUM_THREAD,1,1);
  dim3 block=dim3(1,1,1);
  uint32_t* arr_device, *toadd_device;
  int * sec_device;

  hipMalloc((void**) &arr_device, MAXN*sizeof(uint32_t));
  hipMalloc((void**) &sec_device, (NUM_THREAD+1)*sizeof(int));
  hipMalloc((void**) &toadd_device, (MAXN)*sizeof(uint32_t));
  while (scanf("%d %" PRIu32, &n, &key) == 2) {
    uint32_t sum = 0;
	section[0] = 0;
	section[1] = n/NUM_THREAD;
	for(int i = 2; i < NUM_THREAD; i++){
	  section[i] = section[i-1] + section[1];
	}
	section[NUM_THREAD] = n; // section[3](not include) ~ n

	//hipMemcpy(arr_device, prefix_sum, MAXN*sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(sec_device, section, (NUM_THREAD+1)*sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(toadd_device, toAdd, (MAXN)*sizeof(uint32_t), hipMemcpyHostToDevice);  
  
	add<<<grid,block>>>(sec_device, arr_device, toadd_device, key);
	hipError_t r=hipGetLastError();
	printf("add: %s\n", hipGetErrorString(r));

	hipMemcpy(prefix_sum, arr_device, MAXN*sizeof(uint32_t), hipMemcpyDeviceToHost);		
	
	hipFree(arr_device);
  
	return 0;
  }
