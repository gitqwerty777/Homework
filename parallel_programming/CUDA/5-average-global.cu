#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#define MAXGPU 10
#define BSIDE 16
#define MAXK 40960
#define MAXN 2048

int m, n;
float arr[2*MAXN*MAXN];

__global__ void goNextState(int n, int t, float *arr){
  //global and local
  int i = blockDim.x*blockIdx.x+threadIdx.x+1;
  int j = blockDim.y*blockIdx.y+threadIdx.y+1;
  
  float sum = arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j]
	+ arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j+1]
	+ arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j-1] 
	+ arr[((t+1)%2)*MAXN*MAXN+(i)*MAXN+j+1]
	+ arr[((t+1)%2)*MAXN*MAXN+(i)*MAXN+j-1]
	+ arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j+1]
	+ arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j]
	+ arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j-1];

  arr[((t)%2)*MAXN*MAXN+(i)*MAXN+j] = sum/8;
}

int main() {
  scanf("%d %d", &n, &m);
  for(int i = 1; i <= n; i++)
    for(int j = 1;j <= n; j++)
      arr[i*MAXN+j] = (i-1)*MAXN+(j-1);
  
  //網格和區塊大小設定
  int globalN = n;
  while(globalN % BSIDE)
	globalN++;
  int gridNum = globalN/BSIDE;
  dim3 grid=dim3(n,n,1);
  dim3 block=dim3(1,1,1);

  //配置裝置記憶體
  float* arr_device;
  struct DebugInfo *debuginfo_device;
  hipMalloc((void**) &arr_device, 2*MAXN*MAXN*sizeof(float));
  hipMemcpy(arr_device, arr, 2*MAXN*MAXN*sizeof(float), hipMemcpyHostToDevice);
#ifdef DEBUG
  cudaMalloc((void**) &debuginfo_device, 2*MAXN*MAXN*sizeof(struct DebugInfo));
  cudaMemcpy(debuginfo_device, debugInfo, 2*MAXN*MAXN*sizeof(struct DebugInfo), cudaMemcpyHostToDevice);
#endif

  //呼叫裝置核心.
  for(int t = 1; t <= m; t++){
	goNextState<<<grid,block>>>(n, t, arr_device);
	//測試是否執行成功.	
	hipError_t r=hipGetLastError();
	//printf("gonextstate: %s\n", cudaGetErrorString(r));
	if(r!=0) goto end;
  }

  //下載裝置記憶體內容到主機上.
  hipMemcpy(arr, arr_device, 2*MAXN*MAXN*sizeof(float), hipMemcpyDeviceToHost);
  //下載裝置記憶體內容到主機上.
#ifdef DEBUG  
  cudaMemcpy(debugInfo, debuginfo_device, 2*MAXN*MAXN*sizeof(struct DebugInfo), cudaMemcpyDeviceToHost);
#endif
  
  /*  for(int i = 1; i <= n; i++){
	  for(int j = 1; j <= n; j++){
	  printf("[%d][%d]: globali = %d, globalj = %d, blockidx = %d, blockidy = %d, threadidx = %d, threadidy = %d, globalvalue = %d, count = %d\n", i, j, debugInfo[i*MAXN+j].i, debugInfo[i*MAXN+j].j, debugInfo[i*MAXN+j].blockidx, debugInfo[i*MAXN+j].blockidy, debugInfo[i*MAXN+j].threadidx, debugInfo[i*MAXN+j].threadidy, debugInfo[i*MAXN+j].globalValue, debugInfo[i*MAXN+j].count);
	  }
	  }*/
  
  //顯示內容
  for(int i = 1; i <= n; i++){
    for(int j = 1; j <= n; j++){
      printf("%f ", arr[((m)%2)*MAXN*MAXN+i*MAXN+j]);
    }
    puts("");
  }  

 end:;
  //釋放裝置記憶體.
  hipFree(arr_device);
#ifdef DEBUG
  cudaFree(debuginfo_device);
#endif
  return 0;

}
