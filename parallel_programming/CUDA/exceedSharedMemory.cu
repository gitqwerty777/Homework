
#include <hip/hip_runtime.h>
//索引用到的緒構體
struct Index{
  uint3 block, thread;
};

//核心:把索引寫入裝置記憶體
__global__ void prob_idx_3d(Index* id){

  __shared__ int test[100000];
  
  //計算區塊索引
  int b=(blockIdx.z*gridDim.y+blockIdx.y)*gridDim.x+blockIdx.x;
  //計算執行緒索引
  int t=(threadIdx.z*blockDim.y+threadIdx.y)*blockDim.x+threadIdx.x;
  //計算區塊中包含的執行緒數目
  int n=blockDim.x*blockDim.y*blockDim.z;
  //執行緒在陣列中對應的位置
  int x=b*n+t;

  //每個執行緒寫入自己的區塊和執行緒索引.
  id[x].block=blockIdx;
  id[x].thread=threadIdx;
}
