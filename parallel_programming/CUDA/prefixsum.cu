#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "utils.h"
 
#define MAXN 10000005
#define NUM_THREAD 100
uint32_t prefix_sum[MAXN];
uint32_t toAdd[MAXN];
int section[NUM_THREAD+1];
void output(uint32_t presum[], int n) {
  uint32_t hash = 0;
  for (int i = 1; i <= n; i++)
    hash += presum[i] * i;
  printf("%" PRIu32 "\n", hash);
}

__global__ void add(int section[NUM_THREAD+1], uint32_t prefix_sum[MAXN], uint32_t toAdd[NUM_THREAD], int key){
  int index = blockIdx.x+1;
  int start = section[index]+1;
  int end = section[index+1]+1;
  uint32_t sum = 0;
  for (int i = start; i <= end; i++) {
	sum += ((i << (key&31)) | (i >> (32-(key&31))) + key)^key;
	prefix_sum[i] = sum;
  }
  toAdd[index] = prefix_sum[end];
}

__global__ void add2(int section[NUM_THREAD+1], uint32_t prefix_sum[MAXN], uint32_t toAdd[NUM_THREAD], int key){
  int index = blockIdx.x+1;
  int start = section[index]+1;
  int end = section[index+1];
  for (int i = start; i <= end; i++) {
	prefix_sum[i] += toAdd[index];
	prefix_sum[i] *= i;//output
  }
}


int main() {
  int n;
  uint32_t key;

  dim3 grid=dim3(NUM_THREAD,1,1);
  dim3 block=dim3(1,1,1);
  uint32_t* arr_device, *toadd_device;
  int * sec_device;

  hipMalloc((void**) &arr_device, MAXN*sizeof(uint32_t));
  hipMalloc((void**) &sec_device, (NUM_THREAD+1)*sizeof(int));
  hipMalloc((void**) &toadd_device, (NUM_THREAD)*sizeof(uint32_t));
  while (scanf("%d %" PRIu32, &n, &key) == 2) {
    uint32_t sum = 0;
	section[0] = 0;
	section[1] = n/NUM_THREAD;
	for(int i = 2; i < NUM_THREAD; i++){
	  section[i] = section[i-1] + section[1];
	}
	section[NUM_THREAD] = n; // section[3](not include) ~ n

	//hipMemcpy(arr_device, prefix_sum, MAXN*sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(sec_device, section, (NUM_THREAD+1)*sizeof(int), hipMemcpyHostToDevice);
	//

	add<<<grid,block>>>(sec_device, arr_device, toadd_device, key);
	//hipError_t r=hipGetLastError();
	//printf("add: %s\n", hipGetErrorString(r));


	hipMemcpy(toAdd, toadd_device, MAXN*sizeof(uint32_t), hipMemcpyDeviceToHost);
	
	for(int i = 1; i < NUM_THREAD; i++){
	  toAdd[i] += toAdd[i-1];
	}
	hipMemcpy(toadd_device, toAdd, (MAXN)*sizeof(uint32_t), hipMemcpyHostToDevice);  


   add2<<<grid,block>>>(sec_device, arr_device, toadd_device, key);


   	hipMemcpy(prefix_sum, arr_device, MAXN*sizeof(uint32_t), hipMemcpyDeviceToHost);		

    output(prefix_sum, n);
  }

  hipFree(arr_device);
  hipFree(sec_device);
  hipFree(toadd_device);  
  
  return 0;
}
