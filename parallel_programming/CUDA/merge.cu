#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//索引用到的緒構體
struct Index{
  uint3 block, thread;
};

//核心:把索引寫入裝置記憶體
__global__ void prob_idx_3d(Index* id){

  //計算區塊索引
  int b=(blockIdx.z*gridDim.y+blockIdx.y)*gridDim.x+blockIdx.x;
  //計算執行緒索引
  int t=(threadIdx.z*blockDim.y+threadIdx.y)*blockDim.x+threadIdx.x;
  //計算區塊中包含的執行緒數目
  int n=blockDim.x*blockDim.y*blockDim.z;
  //執行緒在陣列中對應的位置
  int x=b*n+t;

  //每個執行緒寫入自己的區塊和執行緒索引.
  id[x].block=blockIdx;
  id[x].thread=threadIdx;
}

//主函式
int main(){
  //網格和區塊大小設定
  dim3 grid=dim3(4,1,1);
  dim3 block=dim3(2,3,1);
  printf("gridDim  = dim3(%d,%d,%d)\n", grid.x,grid.y,grid.z);
  printf("blockDim = dim3(%d,%d,%d)\n", block.x,block.y,block.z);

  //計算總執行緒數
  int   num=grid.x*grid.y*grid.z*block.x*block.y*block.z;
  printf("total num of threads = %d\n", num);

  //配置主機記憶體 & 清空
  Index* h=(Index*)malloc(num*sizeof(Index));
  memset(h,0,num*sizeof(Index));

  //配置裝置記憶體 & 清空
  Index* d;
  hipMalloc((void**) &d, num*sizeof(Index));
  hipMemcpy(d, h, num*sizeof(Index), hipMemcpyHostToDevice);

  //呼叫裝置核心.
  prob_idx_3d<<<grid,block>>>(d);

  //測試是否執行成功.
  hipError_t r=hipGetLastError();
  printf("prob_idx_3d: %s\n", hipGetErrorString(r));
  if(r!=0) goto end;

  //下載裝置記憶體內容到主機上.
  hipMemcpy(h, d, num*sizeof(Index), hipMemcpyDeviceToHost);

  //顯示內容
  for(int i=0; i<num; i++){
	printf("h[%d]={block:(%d,%d,%d), thread:(%d,%d,%d)}\n", i,
		   h[i].block.x,  h[i].block.y,  h[i].block.z,
		   h[i].thread.x, h[i].thread.y, h[i].thread.z
		   );
  }

 end:;
  //釋放裝置記憶體.
  hipFree(d);
  free(h);

  return 0;
}
