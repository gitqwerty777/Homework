#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define MAXN 1024
int A[MAXN], B[MAXN], C[MAXN];

__global__ void addone(int arr[MAXN]){
  int li = threadIdx.x;
  int index = blockIdx.x*4+li;  

  __shared__ int sharedint[4];

  sharedint[li] = arr[index];
  __syncthreads();
  if(li == 0)
	for(int i = 0; i < 4; i++)
	  sharedint[i] += 1;
  __syncthreads();  
  arr[index] = sharedint[li];
}

__global__ void vectorAdd(int A[MAXN], int B[MAXN], int C[MAXN]){
  int index = blockIdx.x;
  C[index] = A[index]+B[index];
}

dim3 grid1=dim3(MAXN/4,1,1);
dim3 block1=dim3(4,1,1);
dim3 grid=dim3(MAXN,1,1);
dim3 block=dim3(1,1,1);
hipStream_t stream1, stream0, stream2;

void doAddone(int devicei, int* a, hipStream_t* stream){
  hipSetDevice(devicei);
  addone<<<grid1,block1, 0, *stream>>>(a);
  //cudaError_t r=cudaGetLastError();
  //printf("addone : %s\n", cudaGetErrorString(r));	
}

void doAdd(int devicei, int *a, int *b , int *c){
  hipSetDevice(devicei);
  vectorAdd<<<grid,block, 0, stream2>>>(a, b, c);
  //cudaError_t r=cudaGetLastError();
  //printf("add: %s\n", cudaGetErrorString(r));	
}

int main() {
  int numDevs = 0;
  hipGetDeviceCount(&numDevs);
  printf("total %d devices\n", numDevs);

  int *a, *b, *c, *aa, *ba, *ca;

  hipSetDevice(0);
  hipMalloc((void**) &a, MAXN*sizeof(int));
  hipMalloc((void**) &b, MAXN*sizeof(int));
  hipMalloc((void**) &c, MAXN*sizeof(int));

  hipMemcpy(a, A, MAXN*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b, B, MAXN*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(c, C, MAXN*sizeof(int), hipMemcpyHostToDevice);

  hipStreamCreate(&stream0);
  hipStreamCreate(&stream2);  
  
  hipSetDevice(1);
  hipMalloc((void**) &aa, MAXN*sizeof(int));
  hipMalloc((void**) &ba, MAXN*sizeof(int));
  hipMalloc((void**) &ca, MAXN*sizeof(int));

  hipMemcpy(aa, A, MAXN*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ba, B, MAXN*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ca, C, MAXN*sizeof(int), hipMemcpyHostToDevice);  

  hipStreamCreate(&stream1);

  doAddone(0, a, &stream0);  
  doAddone(1, ba, &stream1);
  hipDeviceSynchronize();  
  hipError_t r=hipGetLastError();
  printf("sync: %s\n", hipGetErrorString(r));
  hipMemcpyAsync(b, ba, MAXN*sizeof(int), hipMemcpyDeviceToDevice, stream0);
  doAdd(0, a, b, c);

  hipMemcpy(A, a, MAXN*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(B, ba, MAXN*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(C, c, MAXN*sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < MAXN; i++)
	printf("%d ", A[i]);
  puts("");
  
  for(int i = 0; i < MAXN; i++)
	printf("%d ", B[i]);
  puts("");
  
  for(int i = 0; i < MAXN; i++)
	printf("%d ", C[i]);
  puts("");

  hipFree(a);
  hipFree(b);
  hipFree(c);  
  return 0;
}
