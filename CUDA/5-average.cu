#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#define MAXGPU 10
#define BSIDE 16
#define MAXK 40960
#define MAXN 2048

int m, n;
float arr[2*MAXN*MAXN];

__global__ void goNextState(int n, int t, float *arr){
  //global and local
  int i = blockDim.x*blockIdx.x+threadIdx.x+1;
  int j = blockDim.y*blockIdx.y+threadIdx.y+1;
  int li = threadIdx.x+1;
  int lj = threadIdx.y+1;
  
  __shared__ float sur[BSIDE+2][BSIDE+2];//surroundings

  float original = arr[((t+1)%2)*MAXN*MAXN+(i)*MAXN+j];
  sur[li][lj] = original;
  //debugInfo[i*MAXN+j].globalValue = original;

  if(li == 1){
	sur[0][lj] = arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j];
	if(lj == 1){//corner
	  sur[0][0] = arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j-1];
	} else if(lj == BSIDE || j == n){
	  sur[0][lj+1] = arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j+1];
	}
  }
  if(lj == 1){
	sur[li][0] = arr[((t+1)%2)*MAXN*MAXN+i*MAXN+j-1];
  }
  if(lj == BSIDE || j == n){
	sur[li][lj+1] = arr[((t+1)%2)*MAXN*MAXN+i*MAXN+j+1];
  }    
  if(li == BSIDE || i == n){
	sur[li+1][lj] = arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j];      
	if(lj == 1){
	  sur[li+1][0] = arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j-1];
	} else if(lj == BSIDE || j == n){
	  sur[li+1][lj+1] = arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j+1];
	}
  }

  __syncthreads();

  if(!(i <= n && j <= n)) return;  
  
  //printf("global (%d, %d), local (%d, %d)\n", i, j, li, lj);
  float sum = 0;          
  //use add
  sum += sur[li-1][lj] + 
	+ sur[li-1][lj+1]
	+ sur[li-1][lj-1]
	+ sur[li][lj+1]
	+ sur[li][lj-1]
	+ sur[li+1][lj+1]
	+ sur[li+1][lj]
	+ sur[li+1][lj-1];
  arr[((t)%2)*MAXN*MAXN+(i)*MAXN+j] = sum/8;
}

int main() {
  scanf("%d %d", &n, &m);
  for(int i = 1; i <= n; i++)
    for(int j = 1;j <= n; j++)
      arr[i*MAXN+j] = (i-1)*MAXN+(j-1);
  
  //網格和區塊大小設定
  int globalN = n;
  while(globalN % BSIDE)
	globalN++;
  int gridNum = globalN/BSIDE;
  dim3 grid=dim3(gridNum,gridNum,1);
  dim3 block=dim3(BSIDE,BSIDE,1);

  //配置裝置記憶體
  float* arr_device;
  struct DebugInfo *debuginfo_device;
  hipMalloc((void**) &arr_device, 2*MAXN*MAXN*sizeof(float));
  hipMemcpy(arr_device, arr, 2*MAXN*MAXN*sizeof(float), hipMemcpyHostToDevice);
#ifdef DEBUG
  cudaMalloc((void**) &debuginfo_device, 2*MAXN*MAXN*sizeof(struct DebugInfo));
  cudaMemcpy(debuginfo_device, debugInfo, 2*MAXN*MAXN*sizeof(struct DebugInfo), cudaMemcpyHostToDevice);
#endif

  //呼叫裝置核心.
  for(int t = 1; t <= m; t++){
	goNextState<<<grid,block>>>(n, t, arr_device);
	//測試是否執行成功.	
	hipError_t r=hipGetLastError();
	//printf("gonextstate: %s\n", cudaGetErrorString(r));
	if(r!=0) goto end;
  }

  //下載裝置記憶體內容到主機上.
  hipMemcpy(arr, arr_device, 2*MAXN*MAXN*sizeof(float), hipMemcpyDeviceToHost);
  //下載裝置記憶體內容到主機上.
#ifdef DEBUG  
  cudaMemcpy(debugInfo, debuginfo_device, 2*MAXN*MAXN*sizeof(struct DebugInfo), cudaMemcpyDeviceToHost);
#endif
  
  /*  for(int i = 1; i <= n; i++){
	  for(int j = 1; j <= n; j++){
	  printf("[%d][%d]: globali = %d, globalj = %d, blockidx = %d, blockidy = %d, threadidx = %d, threadidy = %d, globalvalue = %d, count = %d\n", i, j, debugInfo[i*MAXN+j].i, debugInfo[i*MAXN+j].j, debugInfo[i*MAXN+j].blockidx, debugInfo[i*MAXN+j].blockidy, debugInfo[i*MAXN+j].threadidx, debugInfo[i*MAXN+j].threadidy, debugInfo[i*MAXN+j].globalValue, debugInfo[i*MAXN+j].count);
	  }
	  }*/
  
  //顯示內容
  for(int i = 1; i <= n; i++){
    for(int j = 1; j <= n; j++){
      printf("%f ", arr[((m)%2)*MAXN*MAXN+i*MAXN+j]);
    }
    puts("");
  }  

 end:;
  //釋放裝置記憶體.
  hipFree(arr_device);
#ifdef DEBUG
  cudaFree(debuginfo_device);
#endif
  return 0;

}
