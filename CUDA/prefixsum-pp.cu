#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "utils.h"
 
#define MAXN 10000005
#define NUM_THREAD 100
uint32_t prefix_sum[MAXN];
uint32_t toAdd[MAXN];
int section[NUM_THREAD+1];

__global__ void add(int inputs[100], uint32_t outputs[100]){
  int index = blockIdx.x;
  uint32_t key = inputs[index];
  uint32_t sum = 0;
  uint32_t hash = 0;  
  for (int i = 1; i <= MAXN; i++) {
	sum += ((i << (key&31)) | (i >> (32-(key&31))) + key)^key;
    hash += sum * i;	
  }
  outputs[index] = hash;
}

int inputs[100];
uint32_t outputs[100];

int main() {
  int n;
  uint32_t key;

  dim3 grid=dim3(100,1,1);
  dim3 block=dim3(1,1,1);
  int * input_device;
  uint32_t* output_device;

  hipMalloc((void**) &input_device, 100*sizeof(int));
  hipMalloc((void**) &output_device, 100*sizeof(uint32_t));

  int inputcount = 0;
  while (scanf("%d %" PRIu32, &n, &key) == 2) {
	inputs[inputcount++] = key;
  }

  uint32_t sum = 0;
  hipMemcpy(input_device, inputs, (100)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(output_device, outputs, (100)*sizeof(uint32_t), hipMemcpyHostToDevice);  
  add<<<grid,block>>>(input_device, output_device);
  hipError_t r=hipGetLastError();
  hipMemcpy(outputs, output_device, (100)*sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(inputs, input_device, (100)*sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < 100; i++){
	printf("%u\n", outputs[i]);
  }

  hipFree(input_device);
  hipFree(output_device);

  return 0;
}
