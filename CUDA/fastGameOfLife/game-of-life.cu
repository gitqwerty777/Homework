#include "hip/hip_runtime.h"
#define CL_USE_DEPRECATED_OPENCL_2_0_APIS 

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <CL/cl.h>
#define MAXGPU 10
#define BSIDE 8
#define MAXK 40960
#define MAXN 2048

int m, n;
unsigned int arr[2*MAXN*MAXN];

#define MAXN 2048
#define BSIDE 8

__global__ void goNextState(int n, int t, unsigned int *arr){
  //計算區塊索引
  int b=(blockIdx.z*gridDim.y+blockIdx.y)*gridDim.x+blockIdx.x;
  //計算執行緒索引
  int threadnum=(threadIdx.z*blockDim.y+threadIdx.y)*blockDim.x+threadIdx.x;
  //計算區塊中包含的執行緒數目
  int blockthread=blockDim.x*blockDim.y*blockDim.z;
  //執行緒在陣列中對應的位置
  int x=b*n+t;
  //printf("b %d t %d n %d x %d\n", b, threadnum, blockthread, x);
  /*int i = get_global_id(0)+1;
  int j = get_global_id(1)+1;
  int li = get_local_id(0)+1;
  int lj = get_local_id(1)+1;
  
  __shared__ unsigned int sur[BSIDE+2][BSIDE+2];//surroundings
  
  int original = arr[((t+1)%2)*MAXN*MAXN+(i)*MAXN+j];
  sur[li][lj] = original;

  if(!(i <= n && j <= n)) return;  
    if(li == 1){
      sur[0][lj] = arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j];
      if(lj == 1){//corner
	sur[0][0] = arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j-1];
      } else if(lj == BSIDE || j == n){
	sur[0][lj+1] = arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j+1];
      }
    }
    if(lj == 1){
      sur[li][0] = arr[((t+1)%2)*MAXN*MAXN+i*MAXN+j-1];
    }
    if(lj == BSIDE || j == n){
      sur[li][lj+1] = arr[((t+1)%2)*MAXN*MAXN+i*MAXN+j+1];
    }    
    if(li == BSIDE || i == n){
      sur[li+1][lj] = arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j];      
      if(lj == 1){
	sur[li+1][0] = arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j-1];
      } else if(lj == BSIDE || j == n){
	sur[li+1][lj+1] = arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j+1];
      }
    }

  
  barrier(CLK_LOCAL_MEM_FENCE);

  if(!(i <= n && j <= n)) return;

  //printf("global (%d, %d), local (%d, %d)\n", i, j, li, lj);
  int count = 0;          
  //use add
  if(sur[li-1][lj] == '1')
    count++;
  if(sur[li-1][lj+1] == '1')
    count++;
  if(sur[li-1][lj-1] == '1')
    count++;    
  if(sur[li][lj+1] == '1')
    count++;
  if(sur[li][lj-1] == '1')
    count++;
  if(sur[li+1][lj+1] == '1')
    count++;
  if(sur[li+1][lj] == '1')
    count++;    
  if(sur[li+1][lj-1] == '1')
    count++;

  if((original == '1') && !((count == 2) || (count == 3))){
    arr[((t)%2)*MAXN*MAXN+i*MAXN+j] = '0';
    //counts[((t+1)%2)*MAXN*MAXN+i*MAXN+j] = 0;
  } else if((original == '0') && (count == 3)){
    arr[((t)%2)*MAXN*MAXN+i*MAXN+j] = '1';    
    //counts[((t+1)%2)*MAXN*MAXN+i*MAXN+j] = 1;
  } else{
    arr[((t)%2)*MAXN*MAXN+i*MAXN+j] = original;
  }
  */
}

int main() {
  char s[2004];
  scanf("%d %d\n", &n, &m);
  for(int i = 1; i <= n; i++){
    scanf("%s", s);
    for(int j = 1;j <= n; j++)
      arr[i*MAXN+j] = s[j-1];
  }
  
  //網格和區塊大小設定
  dim3 grid=dim3(2,1,1);
  dim3 block=dim3(1,1,1);

  //計算總執行緒數
  int num=grid.x*grid.y*grid.z*block.x*block.y*block.z;

  //配置裝置記憶體
  unsigned int* arr_device;
  hipMalloc((void**) &arr_device, 2*MAXN*MAXN*sizeof(unsigned int));
  hipMemcpy(arr_device, arr, 2*MAXN*MAXN*sizeof(unsigned int), hipMemcpyHostToDevice);

  //呼叫裝置核心.
  for(int t = 0; t < m; t++)
	goNextState<<<grid,block>>>(n, t, arr_device);

  //測試是否執行成功.
  hipError_t r=hipGetLastError();
  printf("gonextstate: %s\n", hipGetErrorString(r));
  if(r!=0) goto end;

  //下載裝置記憶體內容到主機上.
  hipMemcpy(arr, arr_device, 2*MAXN*MAXN*sizeof(unsigned int), hipMemcpyDeviceToHost);
  
  //顯示內容
  for(int i = 1; i <= n; i++){
    for(int j = 1; j <= n; j++){
      putchar((arr[((m)%2)*MAXN*MAXN+i*MAXN+j]==0)?'0':'1');
    }
    puts("");
  }  

 end:;
  //釋放裝置記憶體.
  hipFree(arr_device);
  return 0;

}
