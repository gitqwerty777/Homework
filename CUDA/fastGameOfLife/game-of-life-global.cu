#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#define MAXGPU 10
#define BSIDE 16
#define MAXK 40960
#define MAXN 2048

int m, n;
unsigned int arr[2*MAXN*MAXN];
/*struct DebugInfo{
  int blockidx, blockidy, threadidx, threadidy;
  int i, j, li, lj, globalValue, localValue, count;
  };
  struct DebugInfo debugInfo[2*MAXN*MAXN];
*/

__global__ void goNextState(int n, int t, unsigned int *arr){
  //global and local
  int i = blockDim.x*blockIdx.x+threadIdx.x+1;
  int j = blockDim.y*blockIdx.y+threadIdx.y+1;

#ifdef DEBUG
  debugInfo[i*MAXN+j].i = blockDim.x*blockIdx.x + threadIdx.x;
  debugInfo[i*MAXN+j].j = blockDim.y*blockIdx.y + threadIdx.y;
  debugInfo[i*MAXN+j].blockidx = blockIdx.x;
  debugInfo[i*MAXN+j].blockidy = blockIdx.y;
  debugInfo[i*MAXN+j].threadidx = threadIdx.x;
  debugInfo[i*MAXN+j].threadidy = threadIdx.y;
#endif
  
  //printf("global (%d, %d), local (%d, %d)\n", i, j, li, lj);
  int count = 0;          
  //use add
  if(arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j] == 1)
    count++;
  if(arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j+1] == 1)
    count++;
  if(arr[((t+1)%2)*MAXN*MAXN+(i-1)*MAXN+j-1] == 1)
    count++;    
  if(arr[((t+1)%2)*MAXN*MAXN+(i)*MAXN+j+1] == 1)
    count++;
  if(arr[((t+1)%2)*MAXN*MAXN+(i)*MAXN+j-1] == 1)
    count++;
  if(arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j+1] == 1)
    count++;
  if(arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j] == 1)
    count++;    
  if(arr[((t+1)%2)*MAXN*MAXN+(i+1)*MAXN+j-1] == 1)
    count++;

  //debugInfo[i*MAXN+j].count = count;
  int original = arr[((t+1)%2)*MAXN*MAXN+(i)*MAXN+j];
  
  if((original == 1) && !((count == 2) || (count == 3))){
	arr[((t)%2)*MAXN*MAXN+i*MAXN+j] = 0;
    //counts[((t+1)%2)*MAXN*MAXN+i*MAXN+j] = 0;
  } else if((original == 0) && (count == 3)){
    arr[((t)%2)*MAXN*MAXN+i*MAXN+j] = 1;    
    //counts[((t+1)%2)*MAXN*MAXN+i*MAXN+j] = 1;
  } else{
	arr[((t)%2)*MAXN*MAXN+i*MAXN+j] = original;
  }
  
}

int main() {
  char s[2004];
  scanf("%d %d\n", &n, &m);
  for(int i = 1; i <= n; i++){
    scanf("%s", s);
    for(int j = 1;j <= n; j++)
      arr[i*MAXN+j] = s[j-1]-'0';
  }
  
  //網格和區塊大小設定
  dim3 grid=dim3(n,n,1);
  dim3 block=dim3(1,1,1);

  //計算總執行緒數
  int num=grid.x*grid.y*grid.z*block.x*block.y*block.z;

  /*  for(int i = 1; i <= n; i++){
	  for(int j = 1; j <= n; j++){
	  debugInfo[i*MAXN+j].i = -1;
	  debugInfo[i*MAXN+j].j = -1;
	  debugInfo[i*MAXN+j].li = -1;
	  debugInfo[i*MAXN+j].lj = -1;	  
	  }
	  }*/
  
  //配置裝置記憶體
  unsigned int* arr_device;
  struct DebugInfo *debuginfo_device;
  hipMalloc((void**) &arr_device, 2*MAXN*MAXN*sizeof(unsigned int));
  hipMemcpy(arr_device, arr, 2*MAXN*MAXN*sizeof(unsigned int), hipMemcpyHostToDevice);
#ifdef DEBUG
  cudaMalloc((void**) &debuginfo_device, 2*MAXN*MAXN*sizeof(struct DebugInfo));
  cudaMemcpy(debuginfo_device, debugInfo, 2*MAXN*MAXN*sizeof(struct DebugInfo), cudaMemcpyHostToDevice);
#endif

  //呼叫裝置核心.
  for(int t = 1; t <= m; t++){
	goNextState<<<grid,block>>>(n, t, arr_device);
	//測試是否執行成功.	
	hipError_t r=hipGetLastError();
	//printf("gonextstate: %s\n", cudaGetErrorString(r));
	if(r!=0) goto end;
  }

  //下載裝置記憶體內容到主機上.
  hipMemcpy(arr, arr_device, 2*MAXN*MAXN*sizeof(unsigned int), hipMemcpyDeviceToHost);
  //下載裝置記憶體內容到主機上.
#ifdef DEBUG  
  cudaMemcpy(debugInfo, debuginfo_device, 2*MAXN*MAXN*sizeof(struct DebugInfo), cudaMemcpyDeviceToHost);
#endif
  
  /*  for(int i = 1; i <= n; i++){
	  for(int j = 1; j <= n; j++){
	  printf("[%d][%d]: globali = %d, globalj = %d, blockidx = %d, blockidy = %d, threadidx = %d, threadidy = %d, globalvalue = %d, count = %d\n", i, j, debugInfo[i*MAXN+j].i, debugInfo[i*MAXN+j].j, debugInfo[i*MAXN+j].blockidx, debugInfo[i*MAXN+j].blockidy, debugInfo[i*MAXN+j].threadidx, debugInfo[i*MAXN+j].threadidy, debugInfo[i*MAXN+j].globalValue, debugInfo[i*MAXN+j].count);
	  }
	  }*/
  
  //顯示內容
  for(int i = 1; i <= n; i++){
    for(int j = 1; j <= n; j++){
      putchar((arr[((m)%2)*MAXN*MAXN+i*MAXN+j]==0)?'0':'1');
    }
    puts("");
  }  

 end:;
  //釋放裝置記憶體.
  hipFree(arr_device);
#ifdef DEBUG
  cudaFree(debuginfo_device);
#endif
  return 0;

}
